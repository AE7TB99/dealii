#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2016 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

// Test that Point operations on a CUDA device can be used.

#include <deal.II/base/point.h>

#include "../tests.h"

template <int dim, typename Number>
__global__ void
miscellaneous_kernel()
{
  Point<dim, Number> p_1;
  Point<dim, Number> p_2(Tensor<1, dim, Number>{});
  if (dim == 1)
    Point<dim, Number> p(1.);
  if (dim == 2)
    Point<dim, Number> p(1., 2.);
  if (dim == 3)
    Point<dim, Number> p(1., 2., 3.);

  auto p_3 = Point<dim, Number>::unit_vector(0);

  auto entry_1 = p_1(0);
  p_1(0)       = Number{1.};

  auto p_4 = p_1 + Tensor<1, dim, Number>{};
  auto p_5 = p_1 - Tensor<1, dim, Number>{};
  auto t_1 = p_1 - p_2;
  auto p_6 = -p_3;
  auto p_7 = p_4 / 2.;
  auto p_8 = p_2 * 5.;

  auto s_1 = p_1 * t_1;
  auto s_2 = p_2.square();
  auto s_3 = p_3.distance(p_5);
  auto s_4 = p_4.distance_square(p_1);
}

template <int dim, typename Number>
void
test_gpu()
{
  // Miscellaneous
  miscellaneous_kernel<dim, Number><<<1, 1>>>();
  // Check that the kernel was launched correctly
  AssertCuda(hipGetLastError());
  // Check that there was no problem during the execution of the kernel
  AssertCuda(hipDeviceSynchronize());

  deallog << "OK" << std::endl;
}

int
main()
{
  initlog();

  init_cuda();

  test_gpu<1, double>();
  test_gpu<2, double>();
  test_gpu<3, float>();
  test_gpu<1, float>();
  test_gpu<2, float>();
  test_gpu<3, float>();
}
