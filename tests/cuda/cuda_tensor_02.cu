#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2016 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

// Test operator[] and norm_square of cuda_tensor.

#include <deal.II/base/tensor.h>

#include "../tests.h"

template <int rank, int dim, typename Number>
__global__ void
miscellaneous_kernel()
{
  // constructors
  typename Tensor<rank, dim, Number>::array_type array{};
  Tensor<rank, dim, Number>                      dummy_1(array);
  Tensor<rank, dim, Number>                      dummy_2;
  Tensor<rank, dim, Number>                      dummy_3 = dummy_2;

  // access
  Tensor<rank + 1, dim, Number> initializer_1;
  const auto                    dummy_5 = initializer_1[0];

  // assignment
  dummy_2 = dummy_3;
}

template <int rank, int dim, typename Number>
__global__ void
summation_kernel(Tensor<rank, dim, Number> *t,
                 Tensor<rank, dim, Number> *t1,
                 Tensor<rank, dim, Number> *t2)
{
  *t2 += *t;
  *t1 = *t1 + *t;
}

template <int rank, int dim, typename Number>
__global__ void
subtraction_kernel(Tensor<rank, dim, Number> *t,
                   Tensor<rank, dim, Number> *t1,
                   Tensor<rank, dim, Number> *t2)
{
  *t2 -= *t;
  *t1 = *t1 - *t;
}

template <int rank, int dim, typename Number>
__global__ void
multiplication_kernel(Tensor<rank, dim, Number> *t,
                      Tensor<rank, dim, Number> *t1,
                      Tensor<rank, dim, Number> *t2)
{
  *t1 = *t * Number(2.);
  *t2 = Number(2.) * *t;
  *t *= 2.;
}

template <int rank, int dim, typename Number>
__global__ void
division_kernel(Tensor<rank, dim, Number> *t,
                Tensor<rank, dim, Number> *t1,
                Tensor<rank, dim, Number> *t2)
{
  *t1 = *t / Number(2.);
  *t /= 2.;
  *t2 = *t1;
}

template <int dim, typename Number>
__global__ void init_kernel(Tensor<0, dim, Number> *t)
{
  if (threadIdx.x == 0)
    *t = 1.;
}

template <int dim, typename Number>
__global__ void init_kernel(Tensor<1, dim, Number> *t)
{
  const unsigned int i = threadIdx.x;
  if (i < dim)
    (*t)[i] = i + 1.;
}

template <int dim, typename Number>
__global__ void init_kernel(Tensor<2, dim, Number> *t)
{
  const unsigned int i = threadIdx.y;
  const unsigned int j = threadIdx.x;
  if ((i < dim) && (j < dim))
    (*t)[i][j] = j + i * dim + 1.;
}


template <int rank, int dim, typename Number>
void
test_gpu()
{
  const double tolerance = 1.e-8;

  Tensor<rank, dim, Number> *t_dev;
  Tensor<rank, dim, Number> *t1_dev;
  Tensor<rank, dim, Number> *t2_dev;

  Tensor<rank, dim, Number> t_host;
  Tensor<rank, dim, Number> t1_host;
  Tensor<rank, dim, Number> t2_host;

  Tensor<rank, dim, Number> reference_host;

  // Allocate objects on the device
  hipError_t cuda_error =
    hipMalloc(&t_dev, sizeof(Tensor<rank, dim, Number>));
  AssertCuda(cuda_error);
  cuda_error = hipMalloc(&t1_dev, sizeof(Tensor<rank, dim, Number>));
  AssertCuda(cuda_error);
  cuda_error = hipMalloc(&t2_dev, sizeof(Tensor<rank, dim, Number>));
  AssertCuda(cuda_error);

  // Initialize
  dim3 block_dim(dim, dim);
  init_kernel<<<1, block_dim>>>(t_dev);
  cuda_error = hipMemcpy(&reference_host,
                          t_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);

  // Test multiplication.
  multiplication_kernel<<<1, 1>>>(t_dev, t1_dev, t2_dev);

  cuda_error = hipMemcpy(&t_host,
                          t_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);
  cuda_error = hipMemcpy(&t1_host,
                          t1_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);
  cuda_error = hipMemcpy(&t2_host,
                          t2_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);

  reference_host *= 2;
  AssertThrow((t_host - reference_host).norm() < tolerance, ExcInternalError());
  AssertThrow((t1_host - reference_host).norm() < tolerance,
              ExcInternalError());
  AssertThrow((t2_host - reference_host).norm() < tolerance,
              ExcInternalError());

  deallog << "multiplication OK" << std::endl;

  // Test division.
  division_kernel<<<1, 1>>>(t_dev, t1_dev, t2_dev);
  cuda_error = hipMemcpy(&t_host,
                          t_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);
  cuda_error = hipMemcpy(&t1_host,
                          t1_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);

  reference_host /= 2.;
  AssertThrow((t_host - reference_host).norm() < tolerance, ExcInternalError());
  AssertThrow((t1_host - reference_host).norm() < tolerance,
              ExcInternalError());

  deallog << "division OK" << std::endl;

  // Test summation
  summation_kernel<<<1, 1>>>(t_dev, t1_dev, t2_dev);
  cuda_error = hipMemcpy(&t1_host,
                          t1_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);
  cuda_error = hipMemcpy(&t2_host,
                          t2_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);

  reference_host *= 2.;
  AssertThrow((t1_host - reference_host).norm() < tolerance,
              ExcInternalError());
  AssertThrow((t2_host - reference_host).norm() < tolerance,
              ExcInternalError());


  // Test subtraction
  subtraction_kernel<<<1, 1>>>(t_dev, t1_dev, t2_dev);
  cuda_error = hipMemcpy(&t1_host,
                          t1_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);
  cuda_error = hipMemcpy(&t2_host,
                          t2_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);

  reference_host /= 2.;
  AssertThrow((t1_host - reference_host).norm() < tolerance,
              ExcInternalError());
  AssertThrow((t2_host - reference_host).norm() < tolerance,
              ExcInternalError());

  // Miscellaneous
  miscellaneous_kernel<rank, dim, Number><<<1, 1>>>();

  // Free memory
  cuda_error = hipFree(t_dev);
  AssertCuda(cuda_error);
  cuda_error = hipFree(t1_dev);
  AssertCuda(cuda_error);
  cuda_error = hipFree(t2_dev);
  AssertCuda(cuda_error);
}

int
main()
{
  initlog();

  init_cuda();

  test_gpu<0, 3, double>();
  test_gpu<1, 3, double>();
  test_gpu<2, 3, double>();
  test_gpu<0, 3, float>();
  test_gpu<1, 3, float>();
  test_gpu<2, 3, float>();
}
