#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2019 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------



// check AffineConstraints<double>::set_zero(Vector) for parallel distributed
// vectors

#include <deal.II/base/cuda_size.h>

#include <deal.II/dofs/dof_tools.h>

#include <deal.II/grid/grid_generator.h>

#include <deal.II/lac/la_parallel_vector.h>

#include "../tests.h"


__global__ void
initialize_vector(double *vector, int local_size, int offset)
{
  const int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < local_size)
    vector[index] = 1.0 + index + offset;
}


void
test()
{
  unsigned int myid    = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  unsigned int numproc = Utilities::MPI::n_mpi_processes(MPI_COMM_WORLD);

  IndexSet local_active;
  local_active.set_size(2 * numproc);
  local_active.add_range(myid * numproc, (myid + 1) * numproc);

  LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> v;
  v.reinit(local_active, complete_index_set(2 * numproc), MPI_COMM_WORLD);

  const int n_blocks = 1 + v.size() / CUDAWrappers::block_size;
  initialize_vector<<<n_blocks, CUDAWrappers::block_size>>>(v.get_values(),
                                                            numproc,
                                                            myid * numproc);
  v.compress(VectorOperation::insert);

  AffineConstraints<double> cm;
  cm.add_line(numproc * myid + 1);
  cm.close();

  deallog << "vector before:" << std::endl;
  v.print(deallog.get_file_stream());

  deallog << std::endl;
  deallog << "CM:" << std::endl;
  cm.print(deallog.get_file_stream());

  cm.set_zero(v);

  deallog << "vector after:" << std::endl;
  v.print(deallog.get_file_stream());

  deallog << "OK" << std::endl;
}


int
main(int argc, char *argv[])
{
  Utilities::MPI::MPI_InitFinalize mpi_initialization(argc, argv, 1);
  MPILogInitAll                    log;

  init_cuda();

  test();
  return 0;
}
